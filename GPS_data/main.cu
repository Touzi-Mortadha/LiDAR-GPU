#include "hip/hip_runtime.h"
#include <string>
#include <iostream>
#include <pcap.h>
#include <stdio.h>
#include <time.h>
#include <math.h>
#include <iostream>
#include <fstream>
#include <sstream> 

using namespace std;

#define PI 3.1415926535 



struct point
{
double x; 
double y;
double z; 
double reflectivity;
}; 

//TODO
/*
double conversion(double lon,double lat)
{

}
*/


//TODO
__global__ 
void analyse(u_char* data, point* d_point,double * d_azimuth,int * d_vertical_angle )
{
	
//	extern __shared__ float s_azimuth[];
//	extern __shared__ float s_vertical_angle[];
//	extern int	


//	__syncthreads(); 
	//my magic equation
	int j = threadIdx.x;
	// point on the first byte of each data point
	int pointer = 100*(int)(j/32) + (j%32)*3+46;
	
	int index_alpha = (int)(j/16);
	int index_omega = (int)(j%16);
	//millimeter as unit
	double distance=(double)((data[pointer+1]<<8)+data[pointer])*0.002; 
	

	double v_angle= d_vertical_angle[index_omega]  * PI / 180.0;
	double v_azimuth=d_azimuth[index_alpha] * PI / 180.0;
	
	//printf("%d   alpha :  %d \t%f\n",pointer,data[pointer],distance);
	d_point[j].x= distance*cosf(v_angle)*sinf(v_azimuth);
	d_point[j].y= distance*cosf(v_angle)*cosf(v_azimuth);
	d_point[j].z= distance*sinf(v_angle);
	d_point[j].reflectivity=data[pointer+2];
}



int main(int argc, char *argv[])
{
	string file = "./vlp.pcap";
	ofstream f;
	f.open ("file.csv");
	//declare host table, use it in CPU
	double azimuth_array[24];
	point h_point[384];//the return of coordinate of each point in this array of structure
	int vertical_angle[16]={-15,1,-13,-3,-11,5,-9,7,-7,9,-5,11,-3,13,-1,15};//this table is from datasheet
	// Create a character array using a u_char
	const u_char *data;//data in all packet(1248 bytes) 

	//device array for GPU
	int* d_vertical_angle; 	//the real array that we will allocate it in GPU.
	u_char * d_array ; // data array	
	point * d_point;
	double *d_azimuth;
	
	//array sizes in GPU
	int vertical_angle_bytes = 16 * sizeof(int) ;//size of vertical angle array, need it for allocation in GPU
	const int array_bytes = 1248 * sizeof(u_char);//size of data array, need it for allocation in GPU.
	int point_bytes = 384 * sizeof(point);
	int azimuth_bytes = 24* sizeof(double);

	
	//allocate array in GPU
	hipMalloc(&d_vertical_angle,vertical_angle_bytes) ;
	hipMalloc(&d_azimuth,azimuth_bytes);
	hipMalloc(&d_point,point_bytes); 	
	hipMalloc(&d_array,array_bytes);
	
	//memory copy in CUDA GPU
	hipMemcpy(d_vertical_angle,vertical_angle,vertical_angle_bytes,hipMemcpyHostToDevice); 
	
	

	// Create an char array to hold the error.

	char errbuff[PCAP_ERRBUF_SIZE];

	
	// Step 4 - Open the file and store result in pointer to pcap_t
	pcap_t * pcap = pcap_open_offline(file.c_str(), errbuff);



	// Step 5 - Create a header object
	struct pcap_pkthdr *header;
	
	//variable for time calcultaion
	//clock_t start, end;
	//start clock for calculation time
	//start = clock();
	//Step 6 - Loop through packets and print them to screen
	//pcap_next_ex(pcap, &header, &data) ; 
//while(int returnValue = pcap_next_ex(pcap, &header, &data)>=0)
	while (int returnValue = pcap_next_ex(pcap, &header, &data)>=0)
	{
	//	printf("la langueur du trame est %d \n",header->len);
		//s'il s'agit d'une trame parvenant de LIDAR (taille de packet = 1248)
		if (header->len == 1248)
		{
			//pcap_next_ex(pcap, &header, &data);

		//****************** azimuth calculation ***********//
			pcap_next_ex(pcap, &header, &data);
	
			int k =0;
			for(int i = 0;i<12;i++)
			{
				int j = i*100+44;
				int a = data[j];
				int b = data[j+1];
				azimuth_array[k]=(double)((b<<8)+a)/100.0;
				//printf("%d   %d   %f\n",a,b,tab[k]);
				k+=2;
			}
	
			for(int i=1;i<23;i+=2)
			{
				if(azimuth_array[i+1]<azimuth_array[i-1])
				{			
					azimuth_array[i+1]+=360.0;			
				}
				azimuth_array[i] = azimuth_array[i-1]+ (double)(azimuth_array[i+1]-azimuth_array[i-1])/2.0;
				if(azimuth_array[i]>360.0)
					azimuth_array[i]-=360.0;
			}
			azimuth_array[23]=azimuth_array[22];
		

		//********* END of azimuth calculation ***********//

	
			hipMemcpy(d_array,data,array_bytes,hipMemcpyHostToDevice); 
			hipMemcpy(d_azimuth,azimuth_array,azimuth_bytes,hipMemcpyHostToDevice); 
	
			analyse<<<1,384>>>(d_array,d_point,d_azimuth,d_vertical_angle);
			hipDeviceSynchronize();	
	
			hipMemcpy(h_point,d_point,point_bytes,hipMemcpyDeviceToHost);
			for(int i=0;i<384;i++)
			{
				if((int)h_point[i].x!=0 && (int)h_point[i].y!=0 && (int)h_point[i].z !=0 )
				{f<<h_point[i].x<<','<<h_point[i].y<<","<<h_point[i].z<<endl;}
			//	printf("%f\t%f\t%f\n",h_point[i].x,h_point[i].y,h_point[i].z);}
			}
			//time stamp lidar
			int t1=data[1242];
			int t2=data[1243]<<8;
			int t3=data[1244]<<16;
			int t4=data[1245]<<24;
			//second past the hour
			double time_stamp;
			time_stamp = (t1+t2+t3+t4)/1000000.0;
	//		printf("%f\n",time_stamp);
		}
		//si le packet s'agit d'un packet du GPS
		else if (header->len == 554)
		{
			//data of GPS
			int t1,t2,t3,t4;
			double time_stamp;
			t1=data[240];
			t2=data[241]<<8;
			t3=data[242]<<16;
			t4=data[243]<<24;
			//time stamp : second past the hour
			time_stamp=(t1+t2+t3+t4)/1000000.0;
			//printf("%f\n",time_stamp);
				
			//TODO






			stringstream hh;
			string hex=""; 
			for(int i=248;i<420;i++)
				hh<<std::hex<<data[i];
			hex=hh.str();
			std::cout<<hex<<endl;
			int lat_condition=0,lon_condition=0;
			
			if(hex[25]!=',')
				lat_condition=1;
			string lat = hex.substr(16,11+lat_condition);
			if(hex[37+lat_condition]!=',')
				lon_condition=1;
			string lon = hex.substr(28+lat_condition,11+lon_condition);
			cout<<lat<<" " <<lon <<endl;
			int signe_lat =(lat[lat.length()-1]=='N')?1:-1 ;
			int signe_lon =(lon[lon.length()-1]=='E')?1:-1 ;
			double latitude = signe_lat * stod(lat.substr(0,lat.length()-9))+(double)(stod(lat.substr(2+lat_condition,lat.length()-2)))/60.0;
			double longitude = signe_lon * stod(lon.substr(0,lon.length()-9))+(double)(stod(lon.substr(2+lon_condition,lon.length()-2)))/60.0;
			printf("%.5f\n",latitude);
			printf("%.5f\n",longitude);
/*
			string gps[13];
			istringstream ss(hex);
			int i=0;
			while(getline(ss,gps[i++],','))
			{
				if(ss.peek()==',')
					ss.ignore();
			}
//	    		for(int i=0;i<13;i++)
//				std::cout<<gps[i]<<endl;
			
			std::string::size_type sz;	
			int timestamp = std::stoi(gps[1]);
			//cout<<timestamp<<endl;
			char validity=gps[2][0],c=gps[4][0];
			//cout<<validity<<endl;
			double lat=std::stod(gps[3],&sz);
			double lon=std::stof(gps[5]);
			//printf("lat == %.3f      lon == %.3f\n",lat,lon);
			double speed=std::stod(gps[7]); 
			double course=std::stod(gps[8]) ;
			//cout<<speed<<" "<<course<<endl; 
			string datestamp=gps[9]; 
			//cout<<datestamp<<endl;
			double variation=std::stod(gps[10]);
			//cout<<variation<<endl;
			double X,Y;
			//X=(90-lat)*PI
		*/

		}
	}
	
	//end of clock time
	//end = clock();
	f.close();
	//double time_taken = ((double) (end - start)) / CLOCKS_PER_SEC;	
	//printf("fun took %f seconds to execute \n\n\n\n", time_taken);

}

