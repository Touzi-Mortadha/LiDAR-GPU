#include "hip/hip_runtime.h"
#include <string>
#include <iostream>
#include <pcap.h>
#include <stdio.h>
#include <time.h>
#include <math.h>
#include <iostream>
#include <fstream> 

using namespace std;


#define ROTATION_RESOLUTION  0.01f /**< degrees */
#define DISTANCE_RESOLUTION  0.002f
#define PI 3.1415926535 



struct point
{
double x; 
double y;
double z; 
double reflectivity;
}; 




//TODO
__global__ 
void analyse(u_char* data, point* d_point,double * d_azimuth,int * d_vertical_angle )
{
	
//	extern __shared__ float s_azimuth[];
//	extern __shared__ float s_vertical_angle[];
//	extern int	


//	__syncthreads(); 
	//my magic equation
	int j = threadIdx.x;
	// point on the first byte of each data point
	int pointer = 100*(int)(j/32) + (j%32)*3+46;
	
	int index_alpha = (int)(j/16);
	int index_omega = (int)(j%16);
	//millimeter as unit
	double distance=(double)((data[pointer+1]<<8)+data[pointer])*0.002; 
	

	double v_angle= d_vertical_angle[index_omega]  * PI / 180.0;
	double v_azimuth=d_azimuth[index_alpha] * PI / 180.0;
	
	//printf("%d   alpha :  %d \t%f\n",pointer,data[pointer],distance);
	d_point[j].x= distance*cosf(v_angle)*sinf(v_azimuth);
	d_point[j].y= distance*cosf(v_angle)*cosf(v_azimuth);
	d_point[j].z= distance*sinf(v_angle);
	d_point[j].reflectivity=data[pointer+2];
}



int main(int argc, char *argv[])
{
	string file = "./vlp.pcap";
	ofstream f;
	f.open ("file.csv");
	//declare host table, use it in CPU
	double azimuth_array[24];
	point h_point[384];//the return of coordinate of each point in this array of structure
	int vertical_angle[16]={-15,1,-13,-3,-11,5,-9,7,-7,9,-5,11,-3,13,-1,15};//this table is from datasheet
	// Create a character array using a u_char
	const u_char *data;//data in all packet(1248 bytes) 

	//device array for GPU
	int* d_vertical_angle; 	//the real array that we will allocate it in GPU.
	u_char * d_array ; // data array	
	point * d_point;
	double *d_azimuth;
	
	//array sizes in GPU
	int vertical_angle_bytes = 16 * sizeof(int) ;//size of vertical angle array, need it for allocation in GPU
	const int array_bytes = 1248 * sizeof(u_char);//size of data array, need it for allocation in GPU.
	int point_bytes = 384 * sizeof(point);
	int azimuth_bytes = 24* sizeof(double);

	
	//allocate array in GPU
	hipMalloc(&d_vertical_angle,vertical_angle_bytes) ;
	hipMalloc(&d_azimuth,azimuth_bytes);
	hipMalloc(&d_point,point_bytes); 	
	hipMalloc(&d_array,array_bytes);
	
	//memory copy in CUDA GPU
	hipMemcpy(d_vertical_angle,vertical_angle,vertical_angle_bytes,hipMemcpyHostToDevice); 
	
	//variable for time calcultaion
	clock_t start, end;

	// Create an char array to hold the error.

	char errbuff[PCAP_ERRBUF_SIZE];

	
	// Step 4 - Open the file and store result in pointer to pcap_t
	pcap_t * pcap = pcap_open_offline(file.c_str(), errbuff);



	// Step 5 - Create a header object
	struct pcap_pkthdr *header;
	

	//start clock for calculation time
	start = clock();
	//Step 6 - Loop through packets and print them to screen
	//pcap_next_ex(pcap, &header, &data) ; 

	
	while (int returnValue = pcap_next_ex(pcap, &header, &data) >= 0)
	{
		//pcap_next_ex(pcap, &header, &data);

	//****************** azimuth calculation ***********//
		pcap_next_ex(pcap, &header, &data);
	
		int k =0;
		for(int i = 0;i<12;i++)
		{
		int j = i*100+44;
		int a = data[j];
		int b = data[j+1];
		azimuth_array[k]=(double)((b<<8)+a)/100.0;
		//printf("%d   %d   %f\n",a,b,tab[k]);
		k+=2;
		}
	
		for(int i=1;i<23;i+=2)
		{
			if(azimuth_array[i+1]<azimuth_array[i-1])
			{			
				azimuth_array[i+1]+=360.0;			
			}
			azimuth_array[i] = azimuth_array[i-1]+ (double)(azimuth_array[i+1]-azimuth_array[i-1])/2.0;
			if(azimuth_array[i]>360.0)
				azimuth_array[i]-=360.0;
		}
		azimuth_array[23]=azimuth_array[22];
		
	/*	for(int i=1;i<24;i++)
		{
			printf("%f\n",azimuth_array[i]);
		}
	*/
	//********* END of azimuth calculation ***********//
	

	
		hipMemcpy(d_array,data,array_bytes,hipMemcpyHostToDevice); 
		hipMemcpy(d_azimuth,azimuth_array,azimuth_bytes,hipMemcpyHostToDevice); 
	
		analyse<<<1,384>>>(d_array,d_point,d_azimuth,d_vertical_angle);
		hipDeviceSynchronize();	
	
		hipMemcpy(h_point,d_point,point_bytes,hipMemcpyDeviceToHost);
	
	
		for(int i=0;i<384;i++)
		{
			f<<h_point[i].x<<','<<h_point[i].y<<","<<h_point[i].z<<endl;
			printf("%f\t%f\t%f\n",h_point[i].x,h_point[i].y,h_point[i].z);
		}	
	}
	//end of clock time
	end = clock();
	f.close();
	//double time_taken = ((double) (end - start)) / CLOCKS_PER_SEC;	
//	printf("fun took %f seconds to execute \n\n\n\n", time_taken);

}
